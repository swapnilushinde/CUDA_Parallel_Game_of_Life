// GOL.cpp : Defines the entry point for the console application.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>

__device__  void getNeighbor(int *,int,int,int,int,int *);
__global__ void generation(int *input_whole,int *input,int *rowDim,int *colDim,int *tmp,int *m)
{
	int sum = 0;
	int row,col;
	//int * neighborList = (int *) cudaMalloc(8 * sizeof(int));
	int neighborList[8];
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if((tid+(*m)) < ((*rowDim) * (*colDim)))
	{
	row		= (int) (tid+(*m)) / *colDim;
	col	 	= (int) (tid+(*m)) - (row * (*colDim));
	//*(tmp + 2*tid) = row;
	//*(tmp + 2*tid + 1) = col;
	//*dev_row = row;
	//*dev_col = col;
	getNeighbor(input_whole,row,col, *rowDim,*colDim,neighborList);
	for(int k=0;k<8;k++)
	{
		sum += *(neighborList+k);
	}
	//*dev_sum = sum;
	//tmp[tid] = sum;
	//int offset = (row * (*colDim))+col;
	int offset = (row * (*colDim))+col-(*m);
	if(*(input+offset) == 1)
		{
			//current cell has life.
			if(sum<=1)
				*(tmp+offset) = 0;	// under occupancy or loneyness
			else
			{
				if(sum>=4)
					*(tmp+offset) = 0;	// over occupancy or crowded
				else
					*(tmp+offset) = 1; // 2 or 3 neighbors
			}
		}
		else
		{
			//current cell has no life.
			if(sum == 3)
				*(tmp+offset) = 1;	//exactly 3 neighbors gives new birth.
			else
				*(tmp+offset) = 0;
		}
	}
}

//void generation(int *,int,int);
int main(int argc, char *argv[]) 
{
	int *input;
	int inputSize,sqrt_inputSize_int;
	bool infinite_flag = false;
	char reply;
	//Loop continues till number entered by user is perfect square and > 1.
	do{
		printf("Enter number of element- N*N (should be perfect square) \n");
		scanf("%d",&inputSize);
		sqrt_inputSize_int = sqrt((double)inputSize);
		//printf("%d \n",sqrt_inputSize_int);
	}
	while(sqrt_inputSize_int*sqrt_inputSize_int != inputSize || inputSize == 0 || inputSize == 1);
	// Number of rows and columsn is square root of inputSize.
	int rowDim = sqrt((double)inputSize);
	int colDim = sqrt((double)inputSize);
	//Get # of blocks, #of threads and total generations. (default 1).
	// User can choose to get infinite generations also.
	int B,T,totalGenerations=1;
	printf("Enter # of blocks\n");
	scanf("%d",&B);
	printf("Enter # of threads per block\n");
	scanf("%d",&T);
	printf("Do you want infinite loop('y' or 'n') : ");
	//scanf("%c",&reply);
	getchar();
	reply = getchar();
	printf("%c\n",reply);
	if(reply == 'y')
	{
		infinite_flag = true;	
	}
	else
	{
		printf("Enter # of generations\n");
		scanf("%d",&totalGenerations);
	}
	
	input = (int *) malloc(inputSize*sizeof(int));

	//Set input array elements to either 0 or 1 randomly.
	srand(time(NULL));	//Seed rand function each time differently.
	for(int i=0;i<inputSize;i++)
	{
		//*(input+i) = i+1;
		*(input+i) = rand() % 2 ;
	}
	// testing
	int check[] = {0, 0, 1, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 1, 0, 1, 1,
					0, 0, 1, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 1, 0, 1, 1};
	for(int i=0;i<inputSize;i++)
	{
		//*(input+i) = i+1;
		*(input+i) = check[i];
	}
	// testing
	for(int i=0;i<inputSize;i++)
	{
		printf("%d, ",*(input+i));
	}
	printf("\nprinting in row col wise\n");
	//
	for(int i=0;i<rowDim;i++)
	{
		for(int j=0;j<colDim;j++)
		{
			printf("%d ",*(input+(i*colDim)+j)); // i is [row] and j is [col]
		}
		printf("\n");
	}
	// Set output array to default value 99.
	int * tmp = (int *) malloc((rowDim*colDim)*sizeof(int));
	for(int i=0;i<(rowDim*colDim);i++)
	{
		*(tmp + i) = 99;
	}
	//device pointers to copy data to CUDA device.
	int *dev_input,*dev_tmp, *dev_rowDim,*dev_colDim,*dev_offset;
	//int *dev_row,*dev_col,*dev_sum;
	//int row,col,sum;
	int offset=0;
	
	//Create CUDA event for performance measurement.
	hipEvent_t start,end;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	//Start event recording
	hipEventRecord(start, 0);
	
	hipMalloc((void**)&dev_input,(rowDim*colDim) * sizeof(int));
	hipMalloc((void**)&dev_rowDim,sizeof(int));
	hipMalloc((void**)&dev_colDim,sizeof(int));
	hipMalloc((void**)&dev_tmp,(rowDim*colDim) * sizeof(int));
	//cudaMalloc((void**)&dev_col,sizeof(int));
	//cudaMalloc((void**)&dev_row,sizeof(int));
	hipMalloc((void**)&dev_offset,sizeof(int));
	//cudaMalloc((void**)&dev_sum,sizeof(int));

	hipMemcpy(dev_input, input , (rowDim*colDim)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rowDim, &rowDim , sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_colDim, &colDim , sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_tmp, tmp , (rowDim*colDim)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_offset, &offset , sizeof(int),hipMemcpyHostToDevice);
	
	for(int s=0;s<totalGenerations;s++)
	{
		for(int m = 0;m<(rowDim*colDim);m=m+(B*T))
		{
			offset = m;
			hipMemcpy(dev_offset, &offset , sizeof(int),hipMemcpyHostToDevice);
			generation<<<B,T>>>(dev_input,dev_input+m,dev_rowDim,dev_colDim,dev_tmp+m,dev_offset);
			//cudaMemcpy(&row,dev_row,sizeof(int),cudaMemcpyDeviceToHost);
			//cudaMemcpy(&col,dev_col,sizeof(int),cudaMemcpyDeviceToHost);
			//cudaMemcpy(&sum,dev_sum,sizeof(int),cudaMemcpyDeviceToHost);
			hipMemcpy(tmp,dev_tmp,(rowDim*colDim)*sizeof(int),hipMemcpyDeviceToHost);
			/*for(int i=0;i<(rowDim*colDim);i++)
			{
				printf("%d ",*(tmp+i));
			}*/
			printf("\n");
			//int copyoffset;
			//cudaMemcpy(&copyoffset,dev_offset,sizeof(int),cudaMemcpyDeviceToHost);
			//printf("\n%d %d %d %d\n",row,col,copyoffset,sum);
		}
		printf("Generation %d is \n",s+1);
		hipMemcpy(tmp,dev_tmp,(rowDim*colDim)*sizeof(int),hipMemcpyDeviceToHost);
		for(int i=0;i<(rowDim*colDim);i++)
		{
			printf("%d, ",*(tmp+i));
		}
		printf("\ngrid presentation\n");
		for(int i=0;i<rowDim;i++)
		{
			for(int j=0;j<colDim;j++)
			{
				printf("%d ",*(tmp+(i*colDim)+j));
			}
			printf("\n");
		}

		//printf("\n\n");
		hipMemcpy(dev_input, tmp , (rowDim*colDim)*sizeof(int),hipMemcpyHostToDevice);
		if(infinite_flag)
			totalGenerations = totalGenerations + 1;
	}
	//Stop recording
	hipEventRecord(end, 0);
	//Synchronized to actually record event because cudaEvent Record is asynchronous call
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,start,end);
	
	//Final output display
	printf("\nNumber of blocks used %d\n", B);
	printf("Number of threads used %d\n", T);
	printf("Elapsed time in microseconds %ld\n", long(elapsedTime*1000));
	//
	hipEventDestroy(start);
	hipEventDestroy(end);
	
	return 0;
}

__device__  void getNeighbor(int *input,int row,int col,int rowDim,int colDim, int * neighborList)
{
	//printf("elements is %d \n",*(input+(row*colDim)+col));
	int i=0;
	//right neighbor
	if((col+1) < colDim)
	{
		//printf("right neighbor is %d \n",*(input+(row*colDim)+col+1));
		*(neighborList+i) = *(input+(row*colDim)+col+1);
		i++;
	}
	else
	{
		//printf("right neighbor is %d \n",*(input+(row*colDim)+0));
		*(neighborList+i) = *(input+(row*colDim)+0);
		i++;
	}
	//left neighbor
	if((col-1) >= 0)
	{
		//printf("left neighbor is %d \n",*(input+(row*colDim)+col-1));
		*(neighborList+i) = *(input+(row*colDim)+col-1);
		i++;
	}
	else
	{
		//printf("left neighbor is %d \n",*(input+(row*colDim)+colDim-1));
		*(neighborList+i) = *(input+(row*colDim)+colDim-1);
		i++;
	}
	//top neighbor
	if((row-1) >= 0)
	{
		//printf("top neighbor is %d \n",*(input+((row-1)*colDim)+col));
		*(neighborList+i) = *(input+((row-1)*colDim)+col);
		i++;
		//top right neighbor
		if((col+1) < colDim)
		{
			//printf("top right neighbor is %d \n",*(input+((row-1)*colDim)+col+1));
			*(neighborList+i) = *(input+((row-1)*colDim)+col+1);
			i++;
		}
		else
		{
			//printf("top right neighbor is %d \n",*(input+((row-1)*colDim)+0));
			*(neighborList+i) = *(input+((row-1)*colDim)+0);
			i++;
		}
		//top left neighbor
		if((col-1) >= 0)
		{
			//printf("top left neighbor is %d \n",*(input+((row-1)*colDim)+col-1));
			*(neighborList+i) = *(input+((row-1)*colDim)+col-1);
			i++;
		}
		else
		{
			//printf("top left neighbor is %d \n",*(input+((row-1)*colDim)+colDim-1));
			*(neighborList+i) = *(input+((row-1)*colDim)+colDim-1);
			i++;
		}
	}
	else
	{
		//printf("top neighbor is %d \n",*(input+((rowDim-1)*colDim)+col));
		*(neighborList+i) = *(input+((rowDim-1)*colDim)+col);
		i++;
		//top right neighbor
		if((col+1) < colDim)
		{
			//printf("top right neighbor is %d \n",*(input+((rowDim-1)*colDim)+col+1));
			*(neighborList+i) = *(input+((rowDim-1)*colDim)+col+1);
			i++;
		}
		else
		{
			//printf("top right neighbor is %d \n",*(input+((rowDim-1)*colDim)+0));
			*(neighborList+i) = *(input+((rowDim-1)*colDim)+0);
			i++;
		}
		//top left neighbor
		if((col-1) >= 0)
		{
			//printf("top left neighbor is %d \n",*(input+((rowDim-1)*colDim)+col-1));
			*(neighborList+i) = *(input+((rowDim-1)*colDim)+col-1);
			i++;
		}
		else
		{
			//printf("top left neighbor is %d \n",*(input+((rowDim-1)*colDim)+colDim-1));
			*(neighborList+i) = *(input+((rowDim-1)*colDim)+colDim-1);
			i++;
		}
	}
	//bottom neighbor
	if((row+1) < rowDim)
	{
		//bottom neighbor
		//printf("bottom neighbor is %d \n",*(input+((row+1)*colDim)+col));
		*(neighborList+i) = *(input+((row+1)*colDim)+col);
		i++;
		//bottom right neighbor
		if((col +1 ) < colDim)
		{
			//printf("bottom right neighbor is %d \n",*(input+((row+1)*colDim)+col+1));
			*(neighborList+i) = *(input+((row+1)*colDim)+col+1);
			i++;
		}
		else
		{
			//printf("bottom right neighbor is %d \n",*(input+((row+1)*colDim)+0));
			*(neighborList+i) = *(input+((row+1)*colDim)+0);
			i++;
		}
		//bottom left neighbor
		if((col-1) >= 0)
		{
			//printf("bottom left neighbor is %d \n",*(input+((row+1)*colDim)+col-1));
			*(neighborList+i) = *(input+((row+1)*colDim)+col-1);
			i++;
		}
		else
		{
			//printf("bottom left neighbor is %d \n",*(input+((row+1)*colDim)+colDim-1));
			*(neighborList+i) = *(input+((row+1)*colDim)+colDim-1);
			i++;
		}
	}
	else
	{
		//printf("bottom neighbor is %d \n",*(input+(0*colDim)+col));
		*(neighborList+i) = *(input+(0*colDim)+col);
		i++;
		//bottom right neighbor
		if((col +1 ) < colDim)
		{
			//printf("bottom right neighbor is %d \n",*(input+(0*colDim)+col+1));
			*(neighborList+i) = *(input+(0*colDim)+col+1);
			i++;
		}
		else
		{
			//printf("bottom right neighbor is %d \n",*(input+(0*colDim)+0));
			*(neighborList+i) = *(input+(0*colDim)+0);
			i++;
		}
		//bottom left neighbor
		if((col-1) >= 0)
		{
			//printf("bottom left neighbor is %d \n",*(input+(0*colDim)+col-1));
			*(neighborList+i) = *(input+(0*colDim)+col-1);
			i++;
		}
		else
		{
			//printf("bottom left neighbor is %d \n",*(input+(0*colDim)+colDim-1));
			*(neighborList+i) = *(input+(0*colDim)+colDim-1);
			i++;
		}
	}

	//printf("\n value of i %d \n",i);
}
